#include "hip/hip_runtime.h"
#include <Device/Util/Timer.cuh>
#include "Static/KCore/KCore.cuh"
#include <fstream>

#include <nvToolsExt.h>

#define INSERT 0
#define DELETE 1

// #define NVTX_DEBUG

#define NEW_KCORE
// #define SHELL

// #include <Device/Primitives/CubWrapper.cuh>

using namespace timer;
namespace hornets_nest {

KCore::KCore(HornetGraph &hornet) : 
                        StaticAlgorithm(hornet),
                        vqueue(hornet),
                        // src_equeue(hornet, 4.0f),
                        // dst_equeue(hornet, 4.0f),
                        peel_vqueue(hornet),
                        active_queue(hornet),
                        iter_queue(hornet),
                        load_balancing(hornet)
                        {

    h_copy_csr_off   = new vid_t[hornet.nV() + 1]();
    h_copy_csr_edges = new vid_t[0]();
    
    memset(h_copy_csr_off, 0, (hornet.nV() + 1) * sizeof(vid_t));

    gpu::allocate(vertex_pres, hornet.nV());
    gpu::allocate(vertex_color, hornet.nV());
    gpu::allocate(vertex_subg, hornet.nV());
    gpu::allocate(hd_data().src,    hornet.nE());
    gpu::allocate(hd_data().dst,    hornet.nE());
    gpu::allocate(hd_data().src_tot,    hornet.nE());
    gpu::allocate(hd_data().dst_tot,    hornet.nE());
    gpu::allocate(hd_data().counter, 1);
    gpu::allocate(hd_data().counter_tot, 1);
    gpu::memsetZero(hd_data().counter_tot);  // initialize counter for all edge mapping.
}

KCore::~KCore() {
    gpu::free(vertex_pres);
    gpu::free(vertex_color);
    gpu::free(vertex_subg);
    gpu::free(hd_data().src);
    gpu::free(hd_data().dst);
    gpu::free(hd_data().src_tot);
    gpu::free(hd_data().dst_tot);
    gpu::free(hd_data().counter);
    gpu::free(hd_data().counter_tot);
    delete[] h_copy_csr_off;
    delete[] h_copy_csr_edges;
}

void KCore::set_hcopy(HornetGraph *h_copy_arg) {
    h_copy_ptr = h_copy_arg;
}

struct CheckDeg {
    TwoLevelQueue<vid_t> vqueue;
    TwoLevelQueue<vid_t> peel_vqueue;
    vid_t *vertex_pres;
    vid_t *vertex_color;
    uint32_t peel;

    OPERATOR(Vertex &v) {
        vid_t id = v.id();

        if (vertex_pres[id] && v.degree() <= peel) {
            vqueue.insert(id);
            peel_vqueue.insert(id);
            vertex_pres[id] = 0;
            vertex_color[id] = 1;
        }
    } 
};

struct SetPresent {
    vid_t *vertex_pres;

    OPERATOR(Vertex &v) {
        vid_t id = v.id();
        vertex_pres[id] = 1;
    }
};

struct SetColor {
    vid_t *vertex_color;

    OPERATOR(Vertex &v) {
        vid_t id = v.id();
        vertex_color[id] = 0;
    }
};

struct ClearHCopy {
    HostDeviceVar<KCoreData> hd;
    
    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();

        int spot = atomicAdd(hd().counter, 1);
        hd().src[spot] = src;
        hd().dst[spot] = dst;
    }
};

struct PeelVertices {
    HostDeviceVar<KCoreData> hd;
    vid_t *vertex_color;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();

        #if 0
        if (vertex_color[src] && vertex_color[dst]) {
            if (src < dst) {
                int spot = atomicAdd(hd().counter, 1);
                hd().src[spot] = src;
                hd().dst[spot] = dst;

                int spot_rev = atomicAdd(hd().counter, 1);
                hd().src[spot_rev] = dst;
                hd().dst[spot_rev] = src;
            }
        } else if (vertex_color[src] || vertex_color[dst]) {
            int spot = atomicAdd(hd().counter, 1);
            hd().src[spot] = src;
            hd().dst[spot] = dst;

            int spot_rev = atomicAdd(hd().counter, 1);
            hd().src[spot_rev] = dst;
            hd().dst[spot_rev] = src;
        }
        #endif
        
        int cond1 = vertex_color[src] && vertex_color[dst];
        int cond2 = (cond1==0) && (vertex_color[src] || vertex_color[dst]); 

        if ((cond1 && src < dst) || cond2) {
            int spot = atomicAdd(hd().counter, 1);
            hd().src[spot] = src;
            hd().dst[spot] = dst;

            int spot_rev = atomicAdd(hd().counter, 1);
            hd().src[spot_rev] = dst;
            hd().dst[spot_rev] = src;

	}
        #if 0
        int spot = atomicAdd(hd().counter, 1);
        hd().src[spot] = src;
        hd().dst[spot] = dst;

        int spot_rev = atomicAdd(hd().counter, 1);
        hd().src[spot_rev] = dst;
        hd().dst[spot_rev] = src;
        #endif
    }
};

struct Subgraph {
    HostDeviceVar<KCoreData> hd;
    uint32_t peel_edges;
    vid_t *vertex_subg;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();

        if (src < dst && vertex_subg[dst] == 1) {
            int spot = atomicAdd(hd().counter, 1);
            hd().src[spot] = src;
            hd().dst[spot] = dst;

            int spot_rev = atomicAdd(hd().counter, 1);
            hd().src[spot_rev] = dst;
            hd().dst[spot_rev] = src;

            int spot_tot = atomicAdd(hd().counter_tot, 1);
            // uint32_t spot_tot = peel_edges + spot;
            hd().src_tot[spot_tot] = src;
            hd().dst_tot[spot_tot] = dst;
        }
    }
};

struct SubgraphVertices {
    vid_t *vertex_subg;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        vertex_subg[src] = 1;
    }
};

struct ClearSubgraph {
    vid_t *vertex_subg;

    OPERATOR(Vertex &v) {
        vid_t src = v.id();
        vertex_subg[src] = 0;
    }
};

struct PrintVertices {
    const vid_t *src_ptr;
    const vid_t *dst_ptr;
    int32_t size;

    OPERATOR(Vertex &v) {
        if (v.id() == 0) {
            for (uint32_t i = 0; i < size; i++) {
                // printf("%d ", src_ptr[i]);
                printf("batch_src[%u] = %d; batch_dst[%u] = %d;\n", i, src_ptr[i], i,
                                                                   dst_ptr[i]);
            }
        }
    }
};

void KCore::reset() {
    std::cout << "ran1" << std::endl;
}

void oper_bidirect_batch(HornetGraph &hornet, vid_t *src, vid_t *dst, 
                         int size, uint8_t op) {
#ifdef NVTX_DEBUG
    nvtxRangeId_t id3 = nvtxRangeStartA("batch src construct range");
#endif
    gpu::BatchUpdate batch_update(src, dst, size, gpu::BatchType::DEVICE);
    // batch_update.print();

#ifdef NVTX_DEBUG
    nvtxRangeId_t id4 = nvtxRangeStartA("batch insert/delete range");
#endif
    if (op == DELETE) {
        // Delete edges in the forward direction.
        // hornet.deleteEdgeBatch(batch_update_src);
        // hornet.deleteEdgeBatch(batch_update, gpu::batch_property::IN_PLACE);
        hornet.deleteEdgeBatch(batch_update, gpu::batch_property::IN_PLACE);
        // hornet.deleteEdgeBatch(batch_update);
    } else if (op == INSERT) {
        // Delete edges in the forward direction.
        // hornet.insertEdgeBatch(batch_update_src);
        // hornet.insertEdgeBatch(batch_update, gpu::batch_property::IN_PLACE);
        hornet.insertEdgeBatch(batch_update, gpu::batch_property::IN_PLACE);
        // hornet.insertEdgeBatch(batch_update);
    }
#ifdef NVTX_DEBUG
    nvtxRangeEnd(id4);
#endif
}

void kcores(HornetGraph &hornet, 
            HornetGraph &h_copy,
            TwoLevelQueue<vid_t> &vqueue, 
            HostDeviceVar<KCoreData>& hd, 
            TwoLevelQueue<vid_t> &peel_vqueue,
            load_balancing::VertexBased1 load_balancing,
            uint32_t *max_peel,
            vid_t *vertex_pres,
            vid_t *vertex_subg,
            vid_t *vertex_color,
            uint32_t *ne,
            uint32_t peel_edges) {

    uint32_t peel = 0;
    uint32_t nv = hornet.nV();
    int size = 0;
    
#ifdef NVTX_DEBUG
    nvtxRangeId_t id1 = nvtxRangeStartA("iteration range");
#endif
    while (nv > 0) {
#ifdef NVTX_DEBUG
        nvtxRangeId_t id_init = nvtxRangeStartA("init range");
#endif
        forAllVertices(hornet, SetColor { vertex_color });
        forAllVertices(hornet, CheckDeg { vqueue, peel_vqueue, 
                                          vertex_pres, vertex_color, peel });
#ifdef NVTX_DEBUG
        nvtxRangeEnd(id_init);
#endif
        
        vqueue.swap();
        nv -= vqueue.size();
        
        // vqueue.print();

        if (vqueue.size() > 0) {
            // Find all vertices with degree <= peel.
            gpu::memsetZero(hd().counter);  // reset counter. 

#ifdef NVTX_DEBUG
            nvtxRangeId_t id_peel = nvtxRangeStartA("peel range");
#endif
            forAllEdges(hornet, vqueue, PeelVertices { hd, vertex_color }, 
                        load_balancing); 
#ifdef NVTX_DEBUG
            nvtxRangeEnd(id_peel);
#endif

            hipMemcpy(&size, hd().counter, sizeof(int), hipMemcpyDeviceToHost);

            if (size > 0) {
#ifdef NVTX_DEBUG
                nvtxRangeId_t id2 = nvtxRangeStartA("batch range");
#endif
                oper_bidirect_batch(hornet, hd().src, hd().dst, size, DELETE);
                oper_bidirect_batch(h_copy, hd().src, hd().dst, size, INSERT);
#ifdef NVTX_DEBUG
                nvtxRangeEnd(id2);
#endif
            }

            // *ne -= 2 * size;
            *ne -= size;

            vqueue.clear();
        } else {
            peel++;    
            peel_vqueue.swap();
        }
    }
    *max_peel = peel;

    peel_vqueue.swap();

    forAllEdges(h_copy, peel_vqueue, SubgraphVertices { vertex_subg }, load_balancing);

    gpu::memsetZero(hd().counter);  // reset counter. 
    // forAllEdges(h_copy, peel_vqueue, Subgraph { hd, vertex_subg }, load_balancing);
    forAllEdges(h_copy, peel_vqueue, Subgraph { hd, peel_edges, vertex_subg }, 
                load_balancing);

    forAllVertices(h_copy, ClearSubgraph { vertex_subg });
    
    hipMemcpy(&size, hd().counter, sizeof(int), hipMemcpyDeviceToHost);
    
    if (size > 0) {
        oper_bidirect_batch(h_copy, hd().src, hd().dst, size, DELETE);
        // oper_bidirect_batch(h_copy, batch_update, DELETE);
    }
#ifdef NVTX_DEBUG
    nvtxRangeEnd(id1);
#endif
}

HornetGraph* hornet_copy(HornetGraph &hornet,
                         vid_t *h_copy_csr_off,
                         vid_t *h_copy_csr_edges) {
                         // TwoLevelQueue<vid_t> tot_src_equeue,
                         // TwoLevelQueue<vid_t> tot_dst_equeue) {

    HornetInit hornet_init(hornet.nV(), 0, h_copy_csr_off,
                           h_copy_csr_edges, false);

    HornetGraph *h_copy = new HornetGraph(hornet_init);

    return h_copy;
}

void json_dump(vid_t *src, vid_t *dst, uint32_t *peel, uint32_t peel_edges) {
    std::ofstream output_file;
    output_file.open("output.txt");
    
    output_file << "{\n";
    for (uint32_t i = 0; i < peel_edges; i++) {
        output_file << "\"" << src[i] << "," << dst[i] << "\": " << peel[i];
        if (i < peel_edges - 1) {
            output_file << ",";
        }
        output_file << "\n";
    }
    output_file << "}";
    output_file.close();
}

void KCore::run() {
    omp_set_num_threads(72);
    vid_t *src     = new vid_t[hornet.nE() / 2 + 1];
    vid_t *dst     = new vid_t[hornet.nE() / 2 + 1];
    uint32_t len = hornet.nE() / 2 + 1;
    uint32_t *peel = new uint32_t[hornet.nE() / 2 + 1];
    uint32_t peel_edges = 0;
    uint32_t ne = hornet.nE();
    uint32_t ne_orig = hornet.nE();

    auto pres = vertex_pres;
    auto color = vertex_color;
    auto subg = vertex_subg;
    HornetGraph &h_copy = *h_copy_ptr;
    
    forAllnumV(hornet, [=] __device__ (int i){ pres[i] = 1; } );
    forAllnumV(hornet, [=] __device__ (int i){ subg[i] = 0; } );
    forAllnumV(hornet, [=] __device__ (int i){ color[i] = 0; } );

    // HornetGraph &h_copy = *hornet_copy(hornet, h_copy_csr_off,
    //                                   h_copy_csr_edges);

    gpu::memsetZero(hd_data().counter);  // reset counter. 

    forAllEdges(h_copy, ClearHCopy { hd_data }, load_balancing);
    oper_bidirect_batch(h_copy, hd_data().src, hd_data().dst, hornet.nE(), DELETE);

    hornet.reserveBatchOpResource(hornet.nE(),
                                     gpu::batch_property::IN_PLACE | 
                                     gpu::batch_property::REMOVE_BATCH_DUPLICATE |
                                     gpu::batch_property::REMOVE_CROSS_DUPLICATE);

    h_copy.reserveBatchOpResource(hornet.nE(),
                                     gpu::batch_property::IN_PLACE | 
                                     gpu::batch_property::REMOVE_BATCH_DUPLICATE |
                                     gpu::batch_property::REMOVE_CROSS_DUPLICATE);

    uint32_t iter_count = 0; 
    int size = 0;

    Timer<DEVICE> TM;
    TM.start();
    while (peel_edges < ne_orig / 2) {
        uint32_t max_peel = 0;
        ne = ne_orig - 2 * peel_edges;

        if (iter_count % 2) {
            kcores(h_copy, hornet, vqueue, hd_data, peel_vqueue, 
                   load_balancing, &max_peel, vertex_pres, vertex_subg, 
                   // vertex_color, &ne, peel_edges, batch_update);
                   vertex_color, &ne, peel_edges);
            
            forAllVertices(hornet, SetPresent { vertex_pres });
        } else {
            kcores(hornet, h_copy, vqueue, hd_data, peel_vqueue, 
                   load_balancing, &max_peel, vertex_pres, vertex_subg, 
                   // vertex_color, &ne, peel_edges, batch_update);
                   vertex_color, &ne, peel_edges);

            forAllVertices(h_copy, SetPresent { vertex_pres });
        }

        
        std::cout << "max_peel: " << max_peel << "\n";

        hipMemcpy(&size, hd_data().counter, sizeof(int), 
                   hipMemcpyDeviceToHost);
        size /= 2;

        if (size > 0) {
            #if 0
            hipMemcpy(src + peel_edges, hd_data().src, 
                       size * sizeof(vid_t), hipMemcpyDeviceToHost);

            hipMemcpy(dst + peel_edges, hd_data().dst, 
                       size * sizeof(vid_t), hipMemcpyDeviceToHost);
            #endif

            #pragma omp parallel for
            for (uint32_t i = 0; i < size; i++) {
                peel[peel_edges + i] = max_peel;
            }

            peel_edges += size;
        }

        iter_count++;

        if (peel_edges >= len) {
            std::cout << "ooooops" << std::endl;
            std::cout << "peel_edges " << peel_edges << " len " << len << std::endl;
        }
    }
    TM.stop();
    TM.print("KCore");

    hipMemcpy(src, hd_data().src_tot, 
               peel_edges * sizeof(vid_t), hipMemcpyDeviceToHost);

    hipMemcpy(dst, hd_data().dst_tot, 
                peel_edges * sizeof(vid_t), hipMemcpyDeviceToHost);

    json_dump(src, dst, peel, peel_edges);
}

void KCore::release() {
    std::cout << "ran3" << std::endl;
}
}
