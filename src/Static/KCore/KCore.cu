#include "hip/hip_runtime.h"
#include "Static/KCore/KCore.cuh"
#include <fstream>

#define INSERT 0
#define DELETE 1
// #include <Device/Primitives/CubWrapper.cuh>

namespace hornets_nest {

KCore::KCore(HornetGraph &hornet) : 
                        StaticAlgorithm(hornet),
                        vqueue(hornet),
                        // src_equeue(hornet, 4.0f),
                        // dst_equeue(hornet, 4.0f),
                        peel_vqueue(hornet),
                        load_balancing(hornet) {

    h_copy_csr_off   = new vid_t[hornet.nV() + 1]();
    h_copy_csr_edges = new vid_t[0]();
    
    memset(h_copy_csr_off, 0, (hornet.nV() + 1) * sizeof(vid_t));

    gpu::allocate(vertex_pres, hornet.nV());
    cuMalloc(hd_data().src,    hornet.nE());
    cuMalloc(hd_data().dst,    hornet.nE());
    cuMalloc(hd_data().src_dup,    hornet.nE());
    cuMalloc(hd_data().dst_dup,    hornet.nE());
    // gpu::memsetZero(&(hd_data().counter));
    // hd_data().counter = 0;
    cuMalloc(hd_data().counter, 1);
    // memset(h_copy_csr_edges, 0, hornet.nE() * sizeof(vid_t));
}

KCore::~KCore() {
    gpu::free(vertex_pres);
    // gpu::free(h_copy_csr_off);
    // gpu::free(h_copy_csr_edges);
    // delete[] h_copy_csr_off;
    // delete[] h_copy_csr_edges;
}

struct CheckDeg {
    TwoLevelQueue<vid_t> vqueue;
    TwoLevelQueue<vid_t> peel_vqueue;
    vid_t *vertex_pres;
    uint32_t peel;

    OPERATOR(Vertex &v) {
        vid_t id = v.id();

        if (vertex_pres[id] && v.degree() <= peel) {
            vqueue.insert(id);
            peel_vqueue.insert(id);
            vertex_pres[id] = 0;
        }
    } 
};

struct SetPresent {
    vid_t *vertex_pres;

    OPERATOR(Vertex &v) {
        vid_t id = v.id();
        vertex_pres[id] = 1;
    }
};

struct PeelVertices {
    HostDeviceVar<KCoreData> hd;
    // TwoLevelQueue<vid_t> src_equeue;
    // TwoLevelQueue<vid_t> dst_equeue;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();
        #if 0
        int counter = hd().counter;
        hd().src[counter] = src;
        hd().dst[counter] = dst;
        atomicAdd(&(hd().counter), 1);
        #endif
        // src_equeue.insert(src);
        // dst_equeue.insert(dst);
        int spot = atomicAdd(hd().counter, 1);
        hd().src_dup[spot] = src;
        hd().dst_dup[spot] = dst;
    }
};

struct RemoveDuplicates {
    #if 0
    TwoLevelQueue<vid_t> src_equeue;
    TwoLevelQueue<vid_t> dst_equeue;
    const vid_t *src_ptr;
    const vid_t *dst_ptr;
    int32_t size;
    #endif
    HostDeviceVar<KCoreData> hd;
    int size;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();
        
        uint8_t double_exists = 0;
        if (src < dst) {
            for (uint32_t i = 0; i < size; i++) {
                if (hd().src_dup[i] == dst && hd().dst_dup[i] == src) {
                    double_exists = 1;
                    break;
                }
            }
        }

        if (!double_exists) {
            int spot = atomicAdd(hd().counter, 1);
            hd().src[spot] = src;
            hd().dst[spot] = dst;
            // src_equeue.insert(src);
            // dst_equeue.insert(dst);
        }
    }

};

struct Subgraph {
    // TwoLevelQueue<vid_t> src_equeue;
    // TwoLevelQueue<vid_t> dst_equeue;
    HostDeviceVar<KCoreData> hd;
    const vid_t *peelq_ptr;
    int32_t size;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();

        uint8_t exists = 0;
        if (src < dst) {
            for (uint32_t i = 0; i < size; i++) {
                if (peelq_ptr[i] == dst) {
                    exists = 1;
                    break;
                }
            }
        }

        if (exists){
            // src_equeue.insert(src);
            // dst_equeue.insert(dst);
            int spot = atomicAdd(hd().counter, 1);
            hd().src[spot] = src;
            hd().dst[spot] = dst;
        }
    }
};

struct PrintVertices {
    const vid_t *src_ptr;
    const vid_t *dst_ptr;
    int32_t size;

    OPERATOR(Vertex &v) {
        if (v.id() == 0) {
            for (uint32_t i = 0; i < size; i++) {
                // printf("%d %d\n", src_ptr[i], dst_ptr[i]);
                printf("%d ", src_ptr[i]);
                // printf("batch_src[%d] = %d; batch_dst[%d] = %d;\n", i, src_ptr[i], i, dst_ptr[i]);
            }
        }
    }
};

void KCore::reset() {
    std::cout << "ran1" << std::endl;
}

void oper_bidirect_batch(HornetGraph &hornet, vid_t *src, vid_t *dst, 
                         int size, uint8_t op) {


    #if 0
    std::cout << "oper_og: " << unsigned(op) << "\n";
    hornet.print();
    std::cout << "\n\n";
    #endif

    // Sort src_equeue, dst_equeue by src vertex.
    #if 0
    xlib::CubSortPairs2<vid_t, vid_t>::srun(
                       (vid_t*) src_equeue.device_input_ptr(),
                       (vid_t*) dst_equeue.device_input_ptr(),
                                src_equeue.size(),
                        (vid_t) std::numeric_limits<vid_t>::max(),
                        (vid_t) std::numeric_limits<vid_t>::max());

    gpu::BatchUpdate batch_update_src(
                       (vid_t*) src_equeue.device_input_ptr(),
                       (vid_t*) dst_equeue.device_input_ptr(),
                                src_equeue.size(),
                                gpu::BatchType::DEVICE);
    #endif

    // std::cout << "size " << size << "\n";
    #if 0
    xlib::CubSortPairs2<vid_t, vid_t>::srun(src, dst, size,
                        (vid_t) std::numeric_limits<vid_t>::max(),
                        (vid_t) std::numeric_limits<vid_t>::max());
    #endif

    
    int batch_block = 1024;
    int batch_size = size;

    if (batch_size > batch_block) {
        std::cout << "big batch " << batch_size << "\n";
    }

    vid_t *src_ptr = src;
    vid_t *dst_ptr = dst;

    while (batch_size > 0) {
        int this_size = std::min(batch_size, batch_block);
        gpu::BatchUpdate batch_update_src(src_ptr, dst_ptr, this_size, 
                                          gpu::BatchType::DEVICE);

        #if 0
        if (op == INSERT) {
            std::cout << "sorted by src " << size << std::endl;
            forAllVertices(hornet, PrintVertices { src,
                                                   dst,
                                                   size } );
            std::cout << "\n\n";
            std::cout << std::endl;
        }
        #endif

        if (op == DELETE) {
            #if 0
            hornet.allocateEdgeDeletion(src_equeue.size(), 
                                        gpu::batch_property::IN_PLACE);
            #endif

            // Delete edges in the forward direction.
            hornet.deleteEdgeBatch(batch_update_src);
        } else {
            #if 0
            hornet.allocateEdgeInsertion(src_equeue.size(), 
                                    gpu::batch_property::IN_PLACE);
                                    // gpu::batch_property::REMOVE_CROSS_DUPLICATE);
            #endif

            // Delete edges in the forward direction.
            hornet.insertEdgeBatch(batch_update_src);
        }

        batch_size -= this_size;
        src_ptr += this_size;
        dst_ptr += this_size;
    }

    #if 0
    std::cout << "oper_src: " << unsigned(op) << "\n";
    hornet.print();
    std::cout << "\n\n";
    #endif

    // hornet.print();
    // std::cout << "\n\n";
    // Sort src_equeue, dst_equeue by dst vertex.
    #if 0
    xlib::CubSortPairs2<vid_t, vid_t>::srun(
                       (vid_t*) dst_equeue.device_input_ptr(),
                       (vid_t*) src_equeue.device_input_ptr(),
                                src_equeue.size(),
                        (vid_t) std::numeric_limits<vid_t>::max(),
                        (vid_t) std::numeric_limits<vid_t>::max());

    gpu::BatchUpdate batch_update_dst(
                       (vid_t*) dst_equeue.device_input_ptr(),
                       (vid_t*) src_equeue.device_input_ptr(),
                                src_equeue.size(),
                                gpu::BatchType::DEVICE);
    #endif

    #if 0
    xlib::CubSortPairs2<vid_t, vid_t>::srun(dst, src, size,
                        (vid_t) std::numeric_limits<vid_t>::max(),
                        (vid_t) std::numeric_limits<vid_t>::max());
    #endif

    batch_size = size;
    src_ptr = src;
    dst_ptr = dst;

    while (batch_size > 0) {
        int this_size = std::min(batch_size, batch_block);
        gpu::BatchUpdate batch_update_dst(dst_ptr, src_ptr, this_size, 
                                          gpu::BatchType::DEVICE);

        #if 0
        if (op == DELETE) {
            std::cout << "sorted by dst " << src_equeue.size() << std::endl;
            forAllVertices(hornet, PrintVertices { src_equeue.device_input_ptr(),
                                                   dst_equeue.device_input_ptr(),
                                                   src_equeue.size() } );
            std::cout << "\n\n";
        }
        #endif

        if (op == DELETE) {
            #if 0
            hornet.allocateEdgeDeletion(src_equeue.size(), 
                                        gpu::batch_property::IN_PLACE);
            #endif

            // Delete edges in reverse direction.
            hornet.deleteEdgeBatch(batch_update_dst);
        } else {
            #if 0
            hornet.allocateEdgeInsertion(src_equeue.size(), 
                                    gpu::batch_property::IN_PLACE); 
                                    // gpu::batch_property::REMOVE_CROSS_DUPLICATE);
            #endif

            // Delete edges in reverse direction.
            hornet.insertEdgeBatch(batch_update_dst);
        }

        batch_size -= this_size;
        src_ptr += this_size;
        dst_ptr += this_size;
    }

    #if 0
    std::cout << "oper_dst: " << unsigned(op) << "\n";
    hornet.print();
    std::cout << "\n\n";
    #endif
}

void kcores(HornetGraph &hornet, 
            HornetGraph &h_copy,
            TwoLevelQueue<vid_t> &vqueue, 
            HostDeviceVar<KCoreData>& hd, 
            // TwoLevelQueue<vid_t> &src_equeue,
            // TwoLevelQueue<vid_t> &dst_equeue,
            TwoLevelQueue<vid_t> &peel_vqueue,
            load_balancing::VertexBased1 load_balancing,
            uint32_t *max_peel,
            vid_t *vertex_pres,
            uint32_t *ne) {

    uint32_t peel = 0;
    uint32_t nv = hornet.nV();
    int size = 0;
    // hornet.print();

    // while (*ne > 0) {
    while (nv > 0) {
        forAllVertices(hornet, CheckDeg { vqueue, peel_vqueue, 
                                          vertex_pres, peel });
        
        vqueue.swap();
        nv -= vqueue.size();
        
        // vqueue.print();

        if (vqueue.size() > 0) {
            // Find all vertices with degree <= peel.
            gpu::memsetZero(hd().counter);  // reset counter. 
            // hd().counter = 0;
            forAllEdges(hornet, vqueue, 
                        // PeelVertices { src_equeue, dst_equeue }, 
                        PeelVertices { hd }, 
                        load_balancing); 

            // src_equeue.swap();
            // dst_equeue.swap();

            hipMemcpy(&size, hd().counter, sizeof(int), hipMemcpyDeviceToHost);

            // int size = hd().counter;
            // gpu::memsetZero(hd().counter);  // reset counter. 
            // hd().counter = 0;

            // Remove duplicate edges in src_equeue and dst_equeue
            // (can happen if two vertices in vqueue are neighbors).
            #if 0
            forAllEdges(hornet, vqueue,
                        RemoveDuplicates { src_equeue,
                                           dst_equeue,
                                           src_equeue.device_input_ptr(),
                                           dst_equeue.device_input_ptr(),
                                           src_equeue.size() },
                        load_balancing);
            #endif
            #if 0
            forAllEdges(hornet, vqueue,
                        RemoveDuplicates { hd, size },
                        load_balancing);
            #endif

            // src_equeue.swap();
            // dst_equeue.swap();

            // if (src_equeue.size() > 0) {
            hipMemcpy(&size, hd().counter, sizeof(int), hipMemcpyDeviceToHost);
            if (size > 0) {
                // std::cout << "size: " << src_equeue.size() << std::endl;
                // src_equeue.print();
                // dst_equeue.print();

                forAllVertices(hornet, PrintVertices{ hd().src_dup, hd().dst_dup,
                                                      size });
                std::cout << "\n\n";
                forAllVertices(hornet, PrintVertices{ hd().dst_dup, hd().src_dup, 
                                                      size });
                std::cout << "\n\n";
                //oper_bidirect_batch(hornet, src_equeue, dst_equeue, DELETE);
                // oper_bidirect_batch(hornet, hd().src, hd().dst, size, DELETE);
                // oper_bidirect_batch(h_copy, hd().src, hd().dst, size, INSERT);
                oper_bidirect_batch(hornet, hd().src_dup, hd().dst_dup, size, DELETE);
                oper_bidirect_batch(h_copy, hd().src_dup, hd().dst_dup, size, INSERT);
                h_copy.print();
            }

            // *ne -= 2 * src_equeue.size();
            *ne -= 2 * size;

            // Save vqueue if ne == 0 -- these are vertices in the kcore.
            //if (*ne > 0) {
            vqueue.clear();
            //}
        } else {
            peel++;    
            peel_vqueue.swap();
        }
    }
    *max_peel = peel;
    // std::cout << "peel: " << peel << std::endl;

    peel_vqueue.swap();
    // peel_vqueue.print();

    gpu::memsetZero(hd().counter);  // reset counter. 
    // hd().counter = 0;
    forAllEdges(h_copy, peel_vqueue,
                Subgraph { hd,
                           peel_vqueue.device_input_ptr(),
                           peel_vqueue.size() },
                load_balancing);
    
    // src_equeue.swap();
    // dst_equeue.swap();

    // if (src_equeue.size() > 0) {
    hipMemcpy(&size, hd().counter, sizeof(int), hipMemcpyDeviceToHost);
    if (size > 0) {
        // oper_bidirect_batch(h_copy, src_equeue, dst_equeue, DELETE);
        oper_bidirect_batch(h_copy, hd().src, hd().dst, size, DELETE);
    }
}

HornetGraph* hornet_copy(HornetGraph &hornet,
                         vid_t *h_copy_csr_off,
                         vid_t *h_copy_csr_edges) {
                         // TwoLevelQueue<vid_t> tot_src_equeue,
                         // TwoLevelQueue<vid_t> tot_dst_equeue) {

    HornetInit hornet_init(hornet.nV(), 0, h_copy_csr_off,
                           h_copy_csr_edges, false);

    HornetGraph *h_copy = new HornetGraph(hornet_init);

    return h_copy;
}

void json_dump(vid_t *src, vid_t *dst, uint32_t *peel, uint32_t peel_edges) {
    std::ofstream output_file;
    output_file.open("output.txt");
    
    output_file << "{\n";
    for (uint32_t i = 0; i < peel_edges; i++) {
        output_file << "\"" << src[i] << "," << dst[i] << "\": " << peel[i];
        if (i < peel_edges - 1) {
            output_file << ",";
        }
        output_file << "\n";
    }
    output_file << "}";
    output_file.close();
}

void KCore::run() {
    vid_t *src     = new vid_t[hornet.nE() / 2 + 1];
    vid_t *dst     = new vid_t[hornet.nE() / 2 + 1];
    uint32_t *peel = new uint32_t[hornet.nE() / 2 + 1];
    uint32_t peel_edges = 0;
    uint32_t ne = hornet.nE();
    uint32_t ne_orig = hornet.nE();

    auto pres = vertex_pres;
    
    forAllnumV(hornet, [=] __device__ (int i){ pres[i] = 1; } );

    HornetGraph &h_copy = *hornet_copy(hornet, h_copy_csr_off,
                                       h_copy_csr_edges);

    #if 0
    std::cout << "hornet:\n";
    hornet.print();
    std::cout << "\n\n";

    std::cout << "h_copy:\n";
    h_copy.print();
    std::cout << "\n\n";
    #endif

    uint32_t iter_count = 0; 
    int size = 0;
    while (peel_edges < ne_orig / 2) {
        uint32_t max_peel = 0;
        ne = ne_orig - 2 * peel_edges;

        #if 0
        std::cout << "hornet:\n";
        hornet.print();
        std::cout << "\n\n";

        std::cout << "h_copy:\n";
        h_copy.print();
        std::cout << "\n\n";
        #endif

        if (iter_count % 2) {
            // kcores(h_copy, hornet, vqueue, src_equeue, dst_equeue, 
            //       peel_vqueue, load_balancing, &max_peel, vertex_pres, &ne);
            kcores(h_copy, hornet, vqueue, hd_data, peel_vqueue, load_balancing, 
                   &max_peel, vertex_pres, &ne);
            
            forAllVertices(hornet, SetPresent { vertex_pres });
        } else {
            // kcores(hornet, h_copy, vqueue, src_equeue, dst_equeue, 
            //       peel_vqueue, load_balancing, &max_peel, vertex_pres, &ne);
            kcores(hornet, h_copy, vqueue, hd_data, peel_vqueue, load_balancing, 
                   &max_peel, vertex_pres, &ne);

            forAllVertices(h_copy, SetPresent { vertex_pres });
        }

        #if 0
        std::cout << "hornet:\n";
        hornet.print();
        std::cout << "\n\n";

        std::cout << "h_copy:\n";
        h_copy.print();
        std::cout << "\n\n";
        #endif


        // vqueue.print();
        
        std::cout << "max_peel: " << max_peel << "\n";
        // src_equeue.print();
        // dst_equeue.print();

        // if (src_equeue.size() > 0) {
        hipMemcpy(&size, hd_data().counter, sizeof(int), hipMemcpyDeviceToHost);
        if (size > 0) {
            // hipMemcpy(src + peel_edges, src_equeue.device_input_ptr(), 
            //            src_equeue.size() * sizeof(vid_t), hipMemcpyDeviceToHost);
            hipMemcpy(src + peel_edges, hd_data().src, 
                       size * sizeof(vid_t), hipMemcpyDeviceToHost);

            hipMemcpy(dst + peel_edges, hd_data().dst, 
                       size * sizeof(vid_t), hipMemcpyDeviceToHost);

            // for (uint32_t i = 0; i < src_equeue.size(); i++) {
            for (uint32_t i = 0; i < size; i++) {
                peel[peel_edges + i] = max_peel;
            }

            #if 0
            for (uint32_t i = 0; i < src_equeue.size(); i++) {
                tot_src_equeue.insert(src[peel_edges + i]);
                tot_dst_equeue.insert(dst[peel_edges + i]);
            }
            #endif

            // peel_edges += src_equeue.size();
            peel_edges += size;
        }

        // remove_bidirect_batch(hornet, src_equeue, dst_equeue);
        
        #if 0
        if (iter_count % 2) {
            oper_bidirect_batch(hornet, src_equeue, dst_equeue, DELETE);
        } else {
            oper_bidirect_batch(h_copy, src_equeue, dst_equeue, DELETE);
        }
        #endif


        // src_equeue.clear();
        // dst_equeue.clear();
        iter_count++;
        // h_copy.~Hornet();
    }

    json_dump(src, dst, peel, peel_edges);
}


void KCore::release() {
    std::cout << "ran3" << std::endl;
}
}
