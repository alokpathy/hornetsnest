#include "hip/hip_runtime.h"
#include "Static/KCore/KCore.cuh"
#include <fstream>

#define INSERT 0
#define DELETE 1
// #include <Device/Primitives/CubWrapper.cuh>

namespace hornets_nest {

KCore::KCore(HornetGraph &hornet) : 
                        StaticAlgorithm(hornet),
                        vqueue(hornet),
                        src_equeue(hornet),
                        dst_equeue(hornet),
                        peel_vqueue(hornet),
                        load_balancing(hornet) {

    h_copy_csr_off   = new vid_t[hornet.nV() + 1]();
    h_copy_csr_edges = new vid_t[0]();
    
    memset(h_copy_csr_off, 0, (hornet.nV() + 1) * sizeof(vid_t));

    gpu::allocate(vertex_pres, hornet.nV());
    // memset(h_copy_csr_edges, 0, hornet.nE() * sizeof(vid_t));
}

KCore::~KCore() {
    gpu::free(vertex_pres);
    // gpu::free(h_copy_csr_off);
    // gpu::free(h_copy_csr_edges);
    // delete[] h_copy_csr_off;
    // delete[] h_copy_csr_edges;
}

struct CheckDeg {
    TwoLevelQueue<vid_t> vqueue;
    TwoLevelQueue<vid_t> peel_vqueue;
    vid_t *vertex_pres;
    uint32_t peel;

    OPERATOR(Vertex &v) {
        vid_t id = v.id();

        if (vertex_pres[id] && v.degree() <= peel) {
            vqueue.insert(id);
            peel_vqueue.insert(id);
            vertex_pres[id] = 0;
        }
    } 
};

struct SetPresent {
    vid_t *vertex_pres;

    OPERATOR(Vertex &v) {
        vid_t id = v.id();
        vertex_pres[id] = 1;
    }
};

struct PeelVertices {
    // HostDeviceVar<KCoreData> hd;
    TwoLevelQueue<vid_t> src_equeue;
    TwoLevelQueue<vid_t> dst_equeue;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();
        #if 0
        int counter = hd().counter;
        hd().src[counter] = src;
        hd().dst[counter] = dst;
        atomicAdd(&(hd().counter), 1);
        #endif
        src_equeue.insert(src);
        dst_equeue.insert(dst);
    }
};

struct RemoveDuplicates {
    TwoLevelQueue<vid_t> src_equeue;
    TwoLevelQueue<vid_t> dst_equeue;
    const vid_t *src_ptr;
    const vid_t *dst_ptr;
    int32_t size;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();
        
        uint8_t double_exists = 0;
        if (src < dst) {
            for (uint32_t i = 0; i < size; i++) {
                if (src_ptr[i] == dst && dst_ptr[i] == src) {
                    double_exists = 1;
                    break;
                }
            }
        }

        if (!double_exists) {
            src_equeue.insert(src);
            dst_equeue.insert(dst);
        }
    }

};

struct Subgraph {
    TwoLevelQueue<vid_t> src_equeue;
    TwoLevelQueue<vid_t> dst_equeue;
    const vid_t *peelq_ptr;
    int32_t size;

    OPERATOR(Vertex &v, Edge &e) {
        vid_t src = v.id();
        auto dst = e.dst_id();

        uint8_t exists = 0;
        if (src < dst) {
            for (uint32_t i = 0; i < size; i++) {
                if (peelq_ptr[i] == dst) {
                    exists = 1;
                    break;
                }
            }
        }

        if (exists){
            src_equeue.insert(src);
            dst_equeue.insert(dst);
        }
    }
};

struct PrintVertices {
    const vid_t *src_ptr;
    const vid_t *dst_ptr;
    int32_t size;

    OPERATOR(Vertex &v) {
        if (v.id() == 0) {
            for (uint32_t i = 0; i < size; i++) {
                // printf("%d %d\n", src_ptr[i], dst_ptr[i]);
                printf("batch_src[%d] = %d; batch_dst[%d] = %d;\n", i, src_ptr[i], i, dst_ptr[i]);
            }
        }
    }
};

void KCore::reset() {
    std::cout << "ran1" << std::endl;
}

void oper_bidirect_batch(HornetGraph &hornet,
                           TwoLevelQueue<vid_t> src_equeue,
                           TwoLevelQueue<vid_t> dst_equeue,
                           uint8_t op) {


    #if 0
    std::cout << "oper_og: " << unsigned(op) << "\n";
    hornet.print();
    std::cout << "\n\n";
    #endif

    // Sort src_equeue, dst_equeue by src vertex.
    xlib::CubSortPairs2<vid_t, vid_t>::srun(
                       (vid_t*) src_equeue.device_input_ptr(),
                       (vid_t*) dst_equeue.device_input_ptr(),
                                src_equeue.size(),
                        (vid_t) std::numeric_limits<vid_t>::max(),
                        (vid_t) std::numeric_limits<vid_t>::max());

    gpu::BatchUpdate batch_update_src(
                       (vid_t*) src_equeue.device_input_ptr(),
                       (vid_t*) dst_equeue.device_input_ptr(),
                                src_equeue.size());

    #if 0
    if (op == DELETE) {
        std::cout << "sorted by src " << src_equeue.size() << std::endl;
        forAllVertices(hornet, PrintVertices { src_equeue.device_input_ptr(),
                                               dst_equeue.device_input_ptr(),
                                               src_equeue.size() } );
        std::cout << "\n\n";
    }
    #endif

    if (op == DELETE) {
        #if 0
        hornet.allocateEdgeDeletion(src_equeue.size(), 
                                    gpu::batch_property::IN_PLACE);
        #endif

        // Delete edges in the forward direction.
        hornet.deleteEdgeBatch(batch_update_src);
    } else {
        #if 0
        hornet.allocateEdgeInsertion(src_equeue.size(), 
                                gpu::batch_property::IN_PLACE);
                                // gpu::batch_property::REMOVE_CROSS_DUPLICATE);
        #endif

        // Delete edges in the forward direction.
        hornet.insertEdgeBatch(batch_update_src);
    }

    #if 0
    std::cout << "oper_src: " << unsigned(op) << "\n";
    hornet.print();
    std::cout << "\n\n";
    #endif

    // Sort src_equeue, dst_equeue by dst vertex.
    xlib::CubSortPairs2<vid_t, vid_t>::srun(
                       (vid_t*) dst_equeue.device_input_ptr(),
                       (vid_t*) src_equeue.device_input_ptr(),
                                src_equeue.size(),
                        (vid_t) std::numeric_limits<vid_t>::max(),
                        (vid_t) std::numeric_limits<vid_t>::max());

    gpu::BatchUpdate batch_update_dst(
                       (vid_t*) dst_equeue.device_input_ptr(),
                       (vid_t*) src_equeue.device_input_ptr(),
                                src_equeue.size());

    #if 0
    if (op == DELETE) {
        std::cout << "sorted by dst " << src_equeue.size() << std::endl;
        forAllVertices(hornet, PrintVertices { src_equeue.device_input_ptr(),
                                               dst_equeue.device_input_ptr(),
                                               src_equeue.size() } );
        std::cout << "\n\n";
    }
    #endif

    if (op == DELETE) {
        #if 0
        hornet.allocateEdgeDeletion(src_equeue.size(), 
                                    gpu::batch_property::IN_PLACE);
        #endif

        // Delete edges in reverse direction.
        hornet.deleteEdgeBatch(batch_update_dst);
    } else {
        #if 0
        hornet.allocateEdgeInsertion(src_equeue.size(), 
                                gpu::batch_property::IN_PLACE); 
                                // gpu::batch_property::REMOVE_CROSS_DUPLICATE);
        #endif

        // Delete edges in reverse direction.
        hornet.insertEdgeBatch(batch_update_dst);
    }

    #if 0
    std::cout << "oper_dst: " << unsigned(op) << "\n";
    hornet.print();
    std::cout << "\n\n";
    #endif
}

void kcores(HornetGraph &hornet, 
            HornetGraph &h_copy,
            TwoLevelQueue<vid_t> &vqueue, 
            TwoLevelQueue<vid_t> &src_equeue,
            TwoLevelQueue<vid_t> &dst_equeue,
            TwoLevelQueue<vid_t> &peel_vqueue,
            load_balancing::VertexBased1 load_balancing,
            uint32_t *max_peel,
            vid_t *vertex_pres,
            uint32_t *ne) {

    uint32_t peel = 0;
    uint32_t nv = hornet.nV();
    // hornet.print();

    // while (*ne > 0) {
    while (nv > 0) {
        forAllVertices(hornet, CheckDeg { vqueue, peel_vqueue, 
                                          vertex_pres, peel });
        
        vqueue.swap();
        nv -= vqueue.size();
        
        // vqueue.print();

        if (vqueue.size() > 0) {
            // Find all vertices with degree <= peel.
            forAllEdges(hornet, vqueue, 
                        PeelVertices { src_equeue, dst_equeue }, 
                        load_balancing); 

            src_equeue.swap();
            dst_equeue.swap();

            // Remove duplicate edges in src_equeue and dst_equeue
            // (can happen if two vertices in vqueue are neighbors).
            forAllEdges(hornet, vqueue,
                        RemoveDuplicates { src_equeue,
                                           dst_equeue,
                                           src_equeue.device_input_ptr(),
                                           dst_equeue.device_input_ptr(),
                                           src_equeue.size() },
                        load_balancing);

            src_equeue.swap();
            dst_equeue.swap();


            if (src_equeue.size() > 0) {
                // src_equeue.print();
                // dst_equeue.print();

                oper_bidirect_batch(hornet, src_equeue, dst_equeue, DELETE);
                // hornet.print();
                oper_bidirect_batch(h_copy, src_equeue, dst_equeue, INSERT);
            }

            *ne -= 2 * src_equeue.size();

            // Save vqueue if ne == 0 -- these are vertices in the kcore.
            //if (*ne > 0) {
            vqueue.clear();
            //}
        } else {
            peel++;    
            peel_vqueue.swap();
        }
    }
    *max_peel = peel;

    peel_vqueue.swap();
    peel_vqueue.print();

    forAllEdges(h_copy, peel_vqueue,
                Subgraph { src_equeue,
                           dst_equeue,
                           peel_vqueue.device_input_ptr(),
                           peel_vqueue.size() },
                load_balancing);
    
    src_equeue.swap();
    dst_equeue.swap();

    if (src_equeue.size() > 0) {
        oper_bidirect_batch(h_copy, src_equeue, dst_equeue, DELETE);
    }
}

HornetGraph* hornet_copy(HornetGraph &hornet,
                         vid_t *h_copy_csr_off,
                         vid_t *h_copy_csr_edges) {
                         // TwoLevelQueue<vid_t> tot_src_equeue,
                         // TwoLevelQueue<vid_t> tot_dst_equeue) {

    HornetInit hornet_init(hornet.nV(), 0, h_copy_csr_off,
                           h_copy_csr_edges, false);

    HornetGraph *h_copy = new HornetGraph(hornet_init);

    return h_copy;
}

void json_dump(vid_t *src, vid_t *dst, uint32_t *peel, uint32_t peel_edges) {
    std::ofstream output_file;
    output_file.open("output.txt");
    
    output_file << "{\n";
    for (uint32_t i = 0; i < peel_edges; i++) {
        output_file << "\"" << src[i] << "," << dst[i] << "\": " << peel[i];
        if (i < peel_edges - 1) {
            output_file << ",";
        }
        output_file << "\n";
    }
    output_file << "}";
    output_file.close();
}

void KCore::run() {
    vid_t *src     = new vid_t[hornet.nE() / 2 + 1];
    vid_t *dst     = new vid_t[hornet.nE() / 2 + 1];
    uint32_t *peel = new uint32_t[hornet.nE() / 2 + 1];
    uint32_t peel_edges = 0;
    uint32_t ne = hornet.nE();
    uint32_t ne_orig = hornet.nE();

    auto pres = vertex_pres;
    
    forAllnumV(hornet, [=] __device__ (int i){ pres[i] = 1; } );

    HornetGraph &h_copy = *hornet_copy(hornet, h_copy_csr_off,
                                       h_copy_csr_edges);

    #if 0
    std::cout << "hornet:\n";
    hornet.print();
    std::cout << "\n\n";

    std::cout << "h_copy:\n";
    h_copy.print();
    std::cout << "\n\n";
    #endif

    uint32_t iter_count = 0; 
    while (peel_edges < ne_orig / 2) {
        uint32_t max_peel = 0;
        ne = ne_orig - 2 * peel_edges;

        #if 0
        std::cout << "hornet:\n";
        hornet.print();
        std::cout << "\n\n";

        std::cout << "h_copy:\n";
        h_copy.print();
        std::cout << "\n\n";
        #endif

        if (iter_count % 2) {
            kcores(h_copy, hornet, vqueue, src_equeue, dst_equeue, 
                   peel_vqueue, load_balancing, &max_peel, vertex_pres, &ne);
            
            forAllVertices(hornet, SetPresent { vertex_pres });
        } else {
            kcores(hornet, h_copy, vqueue, src_equeue, dst_equeue, 
                   peel_vqueue, load_balancing, &max_peel, vertex_pres, &ne);

            forAllVertices(h_copy, SetPresent { vertex_pres });
        }

        #if 0
        std::cout << "hornet:\n";
        hornet.print();
        std::cout << "\n\n";

        std::cout << "h_copy:\n";
        h_copy.print();
        std::cout << "\n\n";
        #endif


        // vqueue.print();
        
        std::cout << "max_peel: " << max_peel << "\n";
        src_equeue.print();
        dst_equeue.print();

        if (src_equeue.size() > 0) {
            hipMemcpy(src + peel_edges, src_equeue.device_input_ptr(), 
                       src_equeue.size() * sizeof(vid_t), hipMemcpyDeviceToHost);

            hipMemcpy(dst + peel_edges, dst_equeue.device_input_ptr(), 
                       dst_equeue.size() * sizeof(vid_t), hipMemcpyDeviceToHost);

            for (uint32_t i = 0; i < src_equeue.size(); i++) {
                peel[peel_edges + i] = max_peel;
            }

            #if 0
            for (uint32_t i = 0; i < src_equeue.size(); i++) {
                tot_src_equeue.insert(src[peel_edges + i]);
                tot_dst_equeue.insert(dst[peel_edges + i]);
            }
            #endif

            peel_edges += src_equeue.size();
        }

        // remove_bidirect_batch(hornet, src_equeue, dst_equeue);
        
        #if 0
        if (iter_count % 2) {
            oper_bidirect_batch(hornet, src_equeue, dst_equeue, DELETE);
        } else {
            oper_bidirect_batch(h_copy, src_equeue, dst_equeue, DELETE);
        }
        #endif


        src_equeue.clear();
        dst_equeue.clear();
        iter_count++;
        // h_copy.~Hornet();
    }

    json_dump(src, dst, peel, peel_edges);
}


void KCore::release() {
    std::cout << "ran3" << std::endl;
}
}
